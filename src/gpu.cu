#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include "gpu.cuh"

void display_header()
{
    const int kb = 1024;
    const int mb = kb * kb;
    std::cout << "NBody.GPU" << std::endl << "=========" << std::endl << std::endl;

    std::cout << "CUDA version:   v" << CUDART_VERSION << std::endl;
    //std::cout << "Thrust version: v" << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION << std::endl << std::endl;

    int devCount;
    hipGetDeviceCount(&devCount);
    std::cout << "CUDA Devices: " << std::endl << std::endl;

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        std::cout << i << ": " << props.name << ": " << props.major << "." << props.minor << std::endl;
        std::cout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << std::endl;
        std::cout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << std::endl;
        std::cout << "  Constant memory: " << props.totalConstMem / kb << "kb" << std::endl;
        std::cout << "  Block registers: " << props.regsPerBlock << std::endl << std::endl;

        std::cout << "  Warp size:         " << props.warpSize << std::endl;
        std::cout << "  Threads per block: " << props.maxThreadsPerBlock << std::endl;
        std::cout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << std::endl;
        std::cout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << std::endl;
        std::cout << std::endl;
    }
}

int cuda_get_device(int &n)
{

    hipError_t error_status = hipGetDevice(&n);
    std::cout << "n = " << n << std::endl;
    if (error_status != hipSuccess) {
        std::cout << "[Error]: cuda can't get device. [Cuda error]: "
                  << hipGetErrorString( error_status ) << std::endl;
        return 1;
    }
    return 0;
}

int get_blas_handle(hipblasHandle_t &hd)
{
    static int init[16] = {0};
    static hipblasHandle_t handle[16];
    int n = 0;
    if (cuda_get_device(n)) {
        std::cout << "[Error]: can't get_blas_handle." << std::endl;
        return 1;
    }

    std::cout << "get_blas_handle n = " << n << std::endl;

    if(!init[n]) {
        hipblasStatus_t error_status = hipblasCreate(&handle[n]);
        if (error_status !=  HIPBLAS_STATUS_SUCCESS) {
            std::cout << "[Error]: CUBLAS initialization failed." << std::endl;
            return 1;
        }
        init[n] = 1;
    }
    hd = handle[n];
    return 0;
}

int gpu_malloc(float **data, int len)
{
    hipError_t error_status = hipMalloc((void **)data, len);
    if (error_status != hipSuccess) {
        std::cout << "[Error]: cuda can't malloc. [Cuda error]: "
                  << hipGetErrorString( error_status ) << std::endl;
        return 1;
    }
    return 0;
}

int copy_to_gpu(float *gpu_data, float *cpu_data, int len)
{
    hipError_t error_status = hipMemcpy(gpu_data, cpu_data, len, hipMemcpyHostToDevice);
    if (error_status != hipSuccess) {
        std::cout << "[Error]: cuda can't copy to gpu. [Cuda error]: "
                  << hipGetErrorString( error_status ) << std::endl;
        return 1;
    }
    return 0;
}

int extract_from_gpu(float *cpu_data, float *gpu_data, int len)
{
    hipError_t error_status = hipMemcpy(cpu_data , gpu_data, len, hipMemcpyDeviceToHost);
    if (error_status != hipSuccess) {
        std::cout << "[Error]: cuda can't extract from gpu. [Cuda error]: "
                  << hipGetErrorString( error_status ) << std::endl;
        return 1;
    }
    return 0;
}



