#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "gpu.cuh"

void gemm_gpu(int TA, int TB, int M, int N, int K, float ALPHA,
              float *A_gpu, int lda,
              float *B_gpu, int ldb,
              float BETA,
              float *C_gpu, int ldc)
{

    int init[16] = {0};
    hipblasHandle_t handles[16];
    int i = 0;
    hipGetDevice(&i);
    if(!init[i]) {
        hipblasCreate(&handles[i]);
        init[i] = 1;
    }
    hipblasHandle_t handle = handles[i];

    hipblasSgemm(handle, (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
                                     (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N), N, M, K, &ALPHA, B_gpu, ldb, A_gpu, lda, &BETA, C_gpu, ldc);
}