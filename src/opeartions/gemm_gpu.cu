#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include "gpu.cuh"

void gemm_gpu(int TA, int TB, int M, int N, int K, float ALPHA,
              float *A_gpu, int lda,
              float *B_gpu, int ldb,
              float BETA,
              float *C_gpu, int ldc)
{


    hipblasHandle_t handle;

    if(get_blas_handle(handle)) {
        std::cout << "[Error]: can't run gemm_gpu" << std::endl;
        exit(EXIT_FAILURE);
    }

    hipblasSgemm(handle, (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
                                     (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N), N, M, K, &ALPHA, B_gpu, ldb, A_gpu, lda, &BETA, C_gpu, ldc);

}